#include "hip/hip_runtime.h"
extern "C" {

    #include <stdint.h>
    
    __global__ void loglik(const uint32_t n, const uint32_t dim,
                           const REAL* params, const REAL* x, REAL* res) {
        const uint32_t gid = blockIdx.x * blockDim.x + threadIdx.x;
        if (gid < n) {
            const uint32_t start = dim * gid;
            res[gid] = LOGLIK(params, x + start);
        }
    }

    __global__ void lik(const uint32_t n, const uint32_t dim,
                        const REAL* params, const REAL* x, REAL* res) {
        const uint32_t gid = blockIdx.x * blockDim.x + threadIdx.x;
        if (gid < n) {
            const uint32_t start = dim * gid;
            res[gid] = exp(LOGLIK(params, x + start));
        }
    }
    
    __global__ void evidence_reduce(const uint32_t n, const uint32_t dim,
                                    ACCUMULATOR* x_acc, const REAL* params, const REAL* x) {
        const uint32_t gid = blockIdx.x * blockDim.x + threadIdx.x;
        const uint32_t start = dim * gid;
        const ACCUMULATOR sum = block_reduction_sum((gid < n) ? exp(LOGLIK(params, x + start)) : 0.0f);
        if (threadIdx.x == 0) {
            x_acc[blockIdx.x] = sum;
        }
        
    }
}
