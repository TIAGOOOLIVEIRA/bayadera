#include "hip/hip_runtime.h"
extern "C" {

#include "Random123/philox.h"
#include <float.h>

// =========================== Stretch move ====================================

    int block_reduction_sum_int (const int value) {

        const int local_id = threadIdx.x;

        __shared__ int lacc[WGS];
        lacc[local_id] = value;

        __syncthreads();

        int pacc = value;
        int i = blockDim.x;
        while (i > 0) {
            bool include_odd = (i > ((i >> 1) << 1)) && (local_id == ((i >> 1) - 1));
            i >>= 1;
            if (include_odd) {
                pacc += lacc[local_id + i + 1];
            }
            if (local_id < i) {
                pacc += lacc[local_id + i];
                lacc[local_id] = pacc;
            }
            __syncthreads();
        }

        return lacc[0];
    }

    bool stretch_move(const int K,
                      const int seed,
                      const REAL* params,
                      const REAL* Scompl,
                      REAL* X,
                      REAL* logfn_X,
                      const REAL a,
                      const REAL beta,
                      const int step_counter,
                      const int odd_or_even) {

        // Get the index of this walker Xk
        const int k = blockIdx.x * blockDim.x + threadIdx.x;
        
        // Generate uniform(0,1) floats
        philox4x32_key_t key;
            uint32_t* key_v = key.v;
            key_v[0] = seed;
            key_v[1] = 0xdecafaaa;
            key_v[2] = 0xfacebead;
            key_v[3] = 0x12345678;
            philox4x32_ctr_t cnt;
            uint32_t* cnt_v = cnt.v;
            cnt_v[0] = k;
            cnt_v[1] = step_counter;
            cnt_v[2] = odd_or_even;
            cnt_v[3] = 0xbeeff00d;
            uint32_t* rand_uni = philox4x32(cnt, key).v;
            
            float4 u;
            u.x = u01fpt_oo_32_24(rand_uni[0]);
            u.y = u01fpt_oo_32_24(rand_uni[1]);
            u.z = u01fpt_oo_32_24(rand_uni[2]);
            u.w = u01fpt_oo_32_24(rand_uni[3]);

        // Draw a sample from g(z) using the formula from [Christen 2007]
        const REAL z = (a - 2.0f + 1.0f / a) * u.y * u.y
            + (2.0f * (1.0f - 1.0f / a)) * u.y + 1.0f / a;

        // Draw a walker Xj's index at random from the complementary ensemble S(~i)(t)
        const int j0 = (int)(u.x * K * DIM);
        const int k0 = k * DIM;

        REAL Y[DIM];

        for (int i = 0; i < DIM; i++) {
            const REAL Xji = Scompl[j0 + i];
            Y[i] = Xji + z * (X[k0 + i] - Xji);
        }
        
        const REAL logfn_y = LOGFN(params, Y);
        const REAL q = (isfinite(logfn_y)) ?
            powf(z, DIM - 1) * exp(beta * (logfn_y - logfn_X[k])) : 0.0f;

        const bool accepted = u.z <= q;

        if (accepted) {
            for (int i = 0; i < DIM; i++) {
                X[k0 + i] = Y[i];
            }
            logfn_X[k] = logfn_y;
            return 1;
        } else {
            return 0;
        }
    }

    __global__ void stretch_move_accu(const int n,
                                      const int seed,
                                      const int odd_or_even,
                                      const REAL* params,
                                      const REAL* Scompl,
                                      REAL* X,
                                      REAL* logfn_X,
                                      int* accept,
                                      REAL* means,
                                      const REAL a,
                                      const int step_counter) {

        const int gid = blockIdx.x * blockDim.x + threadIdx.x;
        const bool accepted = (gid < n) &&
            stretch_move(n, seed, params, Scompl, X, logfn_X, a, 1.0f, step_counter, odd_or_even);

        const int accepted_sum = block_reduction_sum_int(accepted ? 1 : 0);
        if (threadIdx.x == 0) {
            accept[blockIdx.x] += accepted_sum;
        }
        
        const int k0 = gid * DIM;
        const int id = n * step_counter * DIM;
        for (int i = 0; i < DIM; i++) {
            const REAL mean_sum = block_reduction_sum((gid < n) ? X[k0 + i] : 0.0f);
            if (threadIdx.x == 0) {
                means[id + i * gridDim.x] += mean_sum;
            }
        }
        
    }

    __global__ void stretch_move_bare(const int n,
                                      const int seed,
                                      const int odd_or_even,
                                      const REAL* params,
                                      const REAL* Scompl,
                                      REAL* X,
                                      REAL* logfn_X,
                                      const REAL a,
                                      const REAL beta,
                                      const int step_counter) {

        const int gid = blockIdx.x * blockDim.x + threadIdx.x;
        if (gid < n) {
            stretch_move(n, seed, params, Scompl, X, logfn_X, a, beta, step_counter, odd_or_even);
        }
    }

// ====================== Walkers initialization ===============================
    __global__ void init_walkers(const int n, const int seed, const REAL2* limits, REAL* xs){

        const int gid = blockIdx.x * blockDim.x + threadIdx.x;
        if (gid < n) {
            const int i = gid * 4;
            const REAL2 limits_m0 = limits[i % DIM];
            const REAL2 limits_m1 = limits[(i + 1) % DIM];
            const REAL2 limits_m2 = limits[(i + 2) % DIM];
            const REAL2 limits_m3 = limits[(i + 3) % DIM];

            // Generate uniform(0,1) floats
                        // Generate uniform(0,1) floats
            philox4x32_key_t key;
            uint32_t* key_v = key.v;
            key_v[0] = seed;
            key_v[1] = 0xdecafaaa;
            key_v[2] = 0xfacebead;
            key_v[3] = 0x12345678;
            philox4x32_ctr_t cnt;
            uint32_t* cnt_v = cnt.v;
            cnt_v[0] = gid;
            cnt_v[1] = 0xf00dcafe;
            cnt_v[2] = 0xdeadbeef;
            cnt_v[3] = 0xbeeff00d;
            uint32_t* rand_uni = philox4x32(cnt, key).v;
            
            float4 u;
            u.x = u01fpt_oo_32_24(rand_uni[0]);
            u.y = u01fpt_oo_32_24(rand_uni[1]);
            u.z = u01fpt_oo_32_24(rand_uni[2]);
            u.w = u01fpt_oo_32_24(rand_uni[3]);
            
            xs[i] = u.x * limits_m0.y + (1.0f - u.x) * limits_m0.x;
            xs[i + 1] = u.y * limits_m1.y + (1.0f - u.y) * limits_m1.x;
            xs[i + 2] = u.z * limits_m2.y + (1.0f - u.z) * limits_m2.x;
            xs[i + 3] = u.w * limits_m3.y + (1.0f - u.w) * limits_m3.x;
        }
    }

    __global__ void logfn(const int n, const REAL* params, const REAL* x, REAL* res) {
        const int gid = blockIdx.x * blockDim.x + threadIdx.x;
        if (gid < n) {
            const int start = DIM * gid;
            REAL px[DIM];
            for (int i = 0; i < DIM; i++) {
                px[i] = x[start + i];
            }
            res[gid] = LOGFN(params, px);
        }
    }

// ======================== Acceptance =========================================

    void block_reduction_sum_ulong (uint64_t* acc, const uint64_t value) {

        const int local_size = blockDim.x;
        const int local_id = threadIdx.x;

        __shared__ uint64_t lacc[WGS];
        lacc[local_id] = value;

        __syncthreads();

        uint64_t pacc = value;
        int i = local_size;
        while (i > 0) {
            bool include_odd = (i > ((i >> 1) << 1)) && (local_id == ((i >> 1) - 1));
            i >>= 1;
            if (include_odd) {
                pacc += lacc[local_id + i + 1];
            }
            if (local_id < i) {
                pacc += lacc[local_id + i];
                lacc[local_id] = pacc;
            }
            __syncthreads();
        }

        if(local_id == 0) {
            acc[blockIdx.x] = pacc;
        }
    }

    __global__ void sum_accept_reduction (const int n, uint64_t* acc) {
        const int gid = blockIdx.x * blockDim.x + threadIdx.x;
        if (gid < n) {
            block_reduction_sum_ulong(acc, acc[gid]);
        }
    }

    __global__ void sum_accept_reduce (const int n, uint64_t* acc, const int* data) {
        const int gid = blockIdx.x * blockDim.x + threadIdx.x;
        if (gid < n) {
            block_reduction_sum_ulong(acc, (uint64_t)data[gid]);
        }
    }

    __global__ void sum_means_vertical (const int m, const int n, REAL* acc, const REAL* data) {
        const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
        const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;
        const int i = n * gid_1 + gid_0;
        const bool valid = (gid_0 < m) && (gid_1 < n);
        const REAL sum = block_reduction_sum_2((valid) ? data[i] : 0.0);
        const bool write = valid && (threadIdx.y == 0);
        if (write) {
            acc[m * blockIdx.y + gid_0] = sum;
        }
    }

    __global__ void subtract_mean (const int dim, const int n, REAL* means, const REAL* mean) {
        const int dim_id = blockIdx.x * blockDim.x + threadIdx.x;
        const int n_id = blockIdx.y * blockDim.y + threadIdx.y;
        const bool valid = (dim_id < dim) && (n_id < n);
        if (valid) {
            means[dim * n_id + dim_id] -= mean[dim_id];
        }
    }

// ======================== Autocovariance =====================================

    REAL2 block_reduction_autocovariance (REAL* c0acc, REAL* dacc, const REAL x2, const REAL xacc) {

        const int local_size = blockDim.x;
        const int local_id = threadIdx.x;

        __shared__ REAL lc0[WGS];
        lc0[local_id] = x2;

        __shared__ REAL ld[WGS];
        ld[local_id] = xacc;

        __syncthreads();

        REAL pc0 = x2;
        REAL pd = xacc;

        int i = local_size;
        while (i > 0) {
            i >>= 1;
            if (local_id < i) {
                pc0 += lc0[local_id + i];
                lc0[local_id] = pc0;
                pd += ld[local_id + i];
                ld[local_id] = pd;
            }
            __syncthreads();
        }

        REAL2 result;
        result.x = lc0[0];
        result.y = ld[0];
        return result;

    }

    __global__ void autocovariance (const int n,
                                    const int lag,
                                    REAL* c0acc,
                                    REAL* dacc,
                                    const REAL* means,
                                    const int imax) {
        
        const int gid = blockIdx.x * blockDim.x + threadIdx.x;
        if (gid < n) {
            const int lid = threadIdx.x;
            const int local_size = blockDim.x;
            const int group_id = blockIdx.x;

            __shared__ REAL local_means[2 * WGS];

            const bool load_lag = (lid < lag) && (gid + local_size < n);
            const bool compute = gid < imax;
            REAL xacc = 0.0f;

            for (int i = 0; i < DIM; i++) {
                const REAL x = means[gid * DIM + i];
                local_means[lid] = x;
                local_means[lid + local_size] = load_lag ? means[gid + local_size] : 0.0f;
                __syncthreads();
                xacc = 0.0f;
                for (int s = 0; s < lag; s++) {
                    xacc += x * local_means[lid + s + 1];
                }
                xacc = compute ? x * x + 2 * xacc : 0.0f;
                const REAL2 sums = block_reduction_autocovariance(c0acc, dacc, compute? x*x : 0.0f, xacc);
                if (lid == 0) {
                    c0acc[group_id * DIM + i] = sums.x;
                    dacc[group_id * DIM + i] = sums.y;
                }

            }
        }
    }
}
