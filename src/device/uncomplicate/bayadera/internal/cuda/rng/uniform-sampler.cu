#include "hip/hip_runtime.h"
extern "C" {

#include "Random123/philox.h"

#ifndef R123_0x1p_23f
#define R123_0x1p_23f 1.1920928955078125E-7f
#endif
    
// Sampling from the uniform distribution
    inline float u01fpt_oo_32_24(uint32_t i) {
        return (0.5f + (i >> 9)) * R123_0x1p_23f;
    }

    __global__ void sample (const int n, const REAL* params, const int seed, float4* x) {
        
        const int gid = blockIdx.x * blockDim.x + threadIdx.x;
        if (gid < n) {
            // Generate uniform(0,1) floats
            philox4x32_key_t key;
            uint32_t* key_v = key.v;
            key_v[0] = seed;
            key_v[1] = 0xdecafaaa;
            key_v[2] = 0xfacebead;
            key_v[3] = 0x12345678;
            philox4x32_ctr_t cnt;
            uint32_t* cnt_v = cnt.v;
            cnt_v[0] = gid;
            cnt_v[1] = 0xf00dcafe;
            cnt_v[2] = 0xdeadbeef;
            cnt_v[3] = 0xbeeff00d;

            const float lower = params[0];
            const float range = params[1] - lower;

            uint32_t* rand_uni = philox4x32(cnt, key).v;
            float4 result;
            result.x = u01fpt_oo_32_24(rand_uni[0]) * range + lower;
            result.y = u01fpt_oo_32_24(rand_uni[1]) * range + lower;
            result.z = u01fpt_oo_32_24(rand_uni[2]) * range + lower;
            result.w = u01fpt_oo_32_24(rand_uni[3]) * range + lower;
            x[gid] = result;
            
        }
    }
}
